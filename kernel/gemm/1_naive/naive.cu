#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "../../../utils.cuh"

#define SIZE 128

__global__ void naive(float* A, float* B, float* C, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx < size && idy < size) {
        float tmp = 0.0f;
        for (int i = 0; i < size; ++i) {
            tmp += A[idy * size + i] * B[i * size + idx];
        }
        C[idy * size + idx] = tmp;
    }
}

int main(int argc, char* argv[]) {
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    float *d_A, *d_B, *d_C;
    auto A = new float[SIZE * SIZE];
    auto B = new float[SIZE * SIZE];
    fill_random(A, SIZE * SIZE);
    fill_random(B, SIZE *SIZE);

    auto C = new float[SIZE * SIZE];
    auto A_size = sizeof(float) * SIZE * SIZE;
    auto B_size = sizeof(float) * SIZE * SIZE;
    auto C_size = sizeof(float) * SIZE * SIZE;
    hipMallocAsync((void**)&d_A, A_size, stream);
    hipMallocAsync((void**)&d_B, B_size, stream);
    hipMallocAsync((void**)&d_C, C_size, stream);
    hipMemcpyAsync(d_A, A, A_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, B, B_size, hipMemcpyHostToDevice, stream);
    dim3 gridDim(CEIL_DIV(SIZE, 32), CEIL_DIV(SIZE, 32));
    dim3 blockDim(32, 32);
    naive<<<gridDim, blockDim, 0, stream>>>(d_A, d_B, d_C, SIZE);
    CUDA_CHECK(hipGetLastError());
    hipMemcpyAsync(C, d_C, C_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    auto C_ref = new float[SIZE * SIZE];
    reference_gemm(SIZE, SIZE, SIZE, A, B, C_ref);
    compare_matrices(C, C_ref, SIZE, SIZE, SIZE);
    
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_ref;
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);
    hipStreamDestroy(stream);
    return 0;
}
