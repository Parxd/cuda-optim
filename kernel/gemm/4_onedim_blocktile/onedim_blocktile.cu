#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "../../../utils.cuh"

#define SIZE 8

template <int block_M, int block_N, int block_K, int thread_M>
__global__ void oneDimBlocktile(int M, int N, int K, float* A, float* B, float* C) {
    // TODO: add assertions for kernel launch parameters
    const int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    // const int global_idy = threadIdx.y + blockIdx.y * blockDim.y;
    __shared__ float A_tile[block_M * block_K];
    __shared__ float B_tile[block_K * block_N];
    int tiles = CEIL_DIV(K, block_K);
    float C_res[thread_M] = {0.0f};  // store intermed. C column & cached B value in TMEM
    for (int tile = 0; tile < tiles; ++tile) {
        A_tile[threadIdx.x * block_K + threadIdx.y] = A[global_idx * K + (block_K * tile + threadIdx.y)];
        B_tile[threadIdx.y * block_N + threadIdx.x] = B[(block_K * tile + threadIdx.y) * N + global_idx];
        __syncthreads();
        for (int k = 0; k < block_K; ++k) {
            float B_tmp = B_tile[k * block_N + threadIdx.x];
            for (int thread_row = 0; thread_row < thread_M; ++thread_row) {
                C_res[thread_row] += A_tile[(thread_M * threadIdx.y + thread_row) * block_K + k] * B_tmp;
            }
        }
        __syncthreads();
    }
    for (int thread_row = 0; thread_row < thread_M; ++thread_row) {
        C[(thread_M * threadIdx.y + thread_row) * N + threadIdx.x] = C_res[thread_row];
    }
}

int main(int argc, char* argv[]) {
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    float *d_A, *d_B, *d_C;
    auto A = new float[SIZE * SIZE];
    auto B = new float[SIZE * SIZE];
    fill_random(A, SIZE * SIZE);
    fill_random(B, SIZE * SIZE);
    
    auto C = new float[SIZE * SIZE];
    auto A_size = sizeof(float) * SIZE * SIZE;
    auto B_size = sizeof(float) * SIZE * SIZE;
    auto C_size = sizeof(float) * SIZE * SIZE;
    hipMallocAsync((void**)&d_A, A_size, stream);
    hipMallocAsync((void**)&d_B, B_size, stream);
    hipMallocAsync((void**)&d_C, C_size, stream);
    hipMemcpyAsync(d_A, A, A_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, B, B_size, hipMemcpyHostToDevice, stream);
    
    constexpr int block_M = 8;
    constexpr int block_N = 8;
    constexpr int block_K = 2;
    constexpr int thread_M = 4;
    dim3 gridDim(CEIL_DIV(SIZE, block_M), CEIL_DIV(SIZE, block_N));
    dim3 blockDim(8, 2);
    oneDimBlocktile<block_M, block_N, block_K, thread_M><<<gridDim, blockDim, 0, stream>>>(SIZE, SIZE, SIZE, d_A, d_B, d_C);
    CUDA_CHECK(hipGetLastError());
    
    hipMemcpyAsync(C, d_C, C_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    auto C_ref = new float[SIZE * SIZE];
    reference_gemm(SIZE, SIZE, SIZE, A, B, C_ref);
    compare_matrices(C, C_ref, SIZE, SIZE);
    
    // print(C, SIZE, SIZE);
    // print(C_ref, SIZE, SIZE);
    
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_ref;
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);
    hipStreamDestroy(stream);
    return 0;
}
